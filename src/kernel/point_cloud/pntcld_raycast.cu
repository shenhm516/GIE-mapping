#include "hip/hip_runtime.h"
#include "pntcld_interfaces.h"
#include <float.h>
#include "ray_cast.h"
#include "par_wave/voxmap_utils.cuh"

namespace PNTCLD_RAYCAST
{

__device__ __forceinline__
bool clearRayLoc(LocMap &loc_map,const int3 &crd, const float &val1, const float &val2, const int &time)
{
    if (loc_map.get_vox_type(crd) != VOXTYPE_OCCUPIED)
    {
        loc_map.atom_add_type_count(crd, -1);
        return true;
    }
    return false;
}

__global__
void FreeKNNCheck(LocMap loc_map, int3* VB_keys_loc_D, bool for_motion_planner, int rbt_r2_grids)
{
    // get the z and y coordinate of the grid we are about to scan
    int3 loc_crd;
    loc_crd.z = blockIdx.x;
    loc_crd.y = threadIdx.x;

    for (loc_crd.x = 0; loc_crd.x < loc_map._local_size.x; ++loc_crd.x)
    {
        int idx_1d = loc_map.coord2idx_local(loc_crd);
        char local_type = loc_map.get_vox_type(loc_crd);
        int cnt = 0;
        if (local_type==VOXTYPE_FREE && 
            loc_crd.x!=0 && (loc_crd.x<loc_map._local_size.x-1) &&
            loc_crd.y!=0 && (loc_crd.y<loc_map._local_size.y-1) &&
            loc_crd.z!=0 && (loc_crd.z<loc_map._local_size.z-1)) {
            for (int i=-1; i<2; i++) {
                for (int j=-1; j<2; j++) {
                    for (int k=-1; k<2; k++) {
                        int3 delta_crd = make_int3(i,j,k);
                        int3 near_crd = loc_crd + delta_crd;
                        char near_type = loc_map.get_vox_type(near_crd);
                        if (near_type==VOXTYPE_FREE || near_type ==VOXTYPE_OCCUPIED) {
                            cnt++;
                        }
                    }
                }
            }

        if (cnt<=26) VB_keys_loc_D[idx_1d] = make_int3(1e6,1e6,1e6); // vox type is unknown
        } else if (local_type==VOXTYPE_FREE) {
            // loc_map.set_vox_type(loc_crd,VOXTYPE_UNKNOWN);
            VB_keys_loc_D[idx_1d] = make_int3(1e6,1e6,1e6); // vox type is unknown
        }
    }
    for (loc_crd.x = 0; loc_crd.x < loc_map._local_size.x; ++loc_crd.x) {
        int idx_1d = loc_map.coord2idx_local(loc_crd);
        if (VB_keys_loc_D[idx_1d].x == 1e6) {
            loc_map.set_vox_type(loc_crd,VOXTYPE_UNKNOWN);
            VB_keys_loc_D[idx_1d] = EMPTY_KEY; // vox type is unknown
        }
    }
}


__global__
void getAllocKeys(LocMap loc_map, int3* VB_keys_loc_D, bool for_motion_planner, int rbt_r2_grids)
{
    // get the z and y coordinate of the grid we are about to scan
    int3 loc_crd;
    loc_crd.z = blockIdx.x;
    loc_crd.y = threadIdx.x;

    for (loc_crd.x = 0; loc_crd.x < loc_map._local_size.x; ++loc_crd.x)
    {
        // set grids around as known and free
        if (for_motion_planner)
        {
            int3 crd2center = loc_crd -loc_map._half_shift;
            if(crd2center.x*crd2center.x + crd2center.y*crd2center.y+crd2center.z*crd2center.z <= rbt_r2_grids)
            {
                loc_map.set_vox_count(loc_crd, -1);
            }

        }

        int idx_1d = loc_map.coord2idx_local(loc_crd);

        int count = loc_map.get_vox_count(loc_crd);

        if (count == 0) {
            // loc_map.set_vox_type(loc_crd,VOXTYPE_UNKNOWN);
            VB_keys_loc_D[idx_1d] = EMPTY_KEY; // vox type is unknown
        }
        else {
            if(count>0) {
                // if (loc_map.get_vox_type(loc_crd)!=VOXTYPE_FREE)
                loc_map.set_vox_type(loc_crd,VOXTYPE_OCCUPIED);
                // else loc_map.set_vox_type(loc_crd,VOXTYPE_FREE);
                // loc_map.set_vox_glb_type(loc_crd,VOXTYPE_OCCUPIED);
            }
            else {
                loc_map.set_vox_type(loc_crd,VOXTYPE_FREE);
                // loc_map.set_vox_glb_type(loc_crd,VOXTYPE_FREE);
            }
            int3 glb_crd = loc_map.loc2glb(loc_crd);
            VB_keys_loc_D[idx_1d] = get_VB_key(glb_crd);
        }
    }
}



__global__
void freeLocObs(LocMap loc_map, float4 *pnt_cld, Projection proj, int pnt_sz, int time)
{
    int ring_id = blockIdx.x;
    int scan_id = threadIdx.x;
    int id = threadIdx.x + blockIdx.x *blockDim.x;

    if(id >= pnt_sz) return;

    float3 glb_pos = proj.L2G*make_float3(pnt_cld[id].x, pnt_cld[id].y, pnt_cld[id].z);

    RAY::rayCastLoc(loc_map, proj.origin,  glb_pos, time, 0.707f*loc_map._local_size.x*loc_map._voxel_width, &clearRayLoc);
}


__global__
void registerLocObs(LocMap loc_map, float4 *pnt_cld, Projection proj,  int pnt_sz, int time)
{
    int ring_id = blockIdx.x;
    int scan_id = threadIdx.x;
    int id = threadIdx.x + blockIdx.x *blockDim.x;

    if(id >= pnt_sz) return;

    float3 glb_pos = proj.L2G*make_float3(pnt_cld[id].x, pnt_cld[id].y, pnt_cld[id].z);
    // if (glb_pos.z >= loc_map._update_min_h && glb_pos.z <= loc_map._update_max_h)
    {
        int3 glb_crd = loc_map.pos2coord(glb_pos);
        int3 loc_crd = loc_map.glb2loc(glb_crd);

        // if (loc_map.get_vox_glb_type(loc_crd)==VOXTYPE_FREE) {
        //     loc_map.set_vox_type(loc_crd,VOXTYPE_DYN);
        //     pnt_cld[id].w = 1;
        // }
        // else {
        loc_map.set_vox_type(loc_crd,VOXTYPE_OCCUPIED);
        loc_map.atom_add_type_count(loc_crd,1);
        // }
    }
}

__global__
void registerLocDyn(LocMap loc_map, float4 *pnt_cld, Projection proj, Projection proj_prev, int pnt_sz, int time)
{
    int ring_id = blockIdx.x;
    int scan_id = threadIdx.x;
    int id = threadIdx.x + blockIdx.x *blockDim.x;

    if(id >= pnt_sz) return;
    float3 glb_pos = proj.L2G*make_float3(pnt_cld[id].x, pnt_cld[id].y, pnt_cld[id].z);
    int3 glb_crd = loc_map.pos2coord(glb_pos);
    // int3 loc_crd = loc_map.glb2loc(glb_crd);
    int3 pivot = loc_map.pos2coord(proj_prev.origin);
    pivot.x -= loc_map._local_size.x/2;
    pivot.y -= loc_map._local_size.y/2;
    pivot.z -= loc_map._local_size.z/2;
    int3 loc_crd = glb_crd-pivot;

    if (loc_map.get_vox_glb_type(loc_crd)==VOXTYPE_FREE) {
        pnt_cld[id].w = 1;
    }
}


void localOGMKernels(LocMap* loc_map, float4 *pnt_cld, Projection proj, Projection proj_prev, PntcldParam param,
                     int3* VB_keys_loc_D, int time, bool for_motion_planner, int rbt_r2_grids)
{
    // Register the point clouds
    registerLocObs<<<param.valid_pnt_count/256+1, 256>>>(*loc_map,pnt_cld,proj,param.valid_pnt_count,time);

    // Free the empty areas
    freeLocObs<<<param.valid_pnt_count/256+1, 256>>>(*loc_map,pnt_cld,proj,param.valid_pnt_count,time);

    const int gridSize = loc_map->_local_size.z;
    const int blkSize = loc_map->_local_size.y;
    getAllocKeys<<<gridSize,blkSize>>>(*loc_map, VB_keys_loc_D, for_motion_planner, rbt_r2_grids);
    FreeKNNCheck<<<gridSize,blkSize>>>(*loc_map, VB_keys_loc_D, for_motion_planner, rbt_r2_grids);
    registerLocDyn<<<param.valid_pnt_count/256+1, 256>>>(*loc_map,pnt_cld,proj,proj_prev,param.valid_pnt_count,time);
}
}
